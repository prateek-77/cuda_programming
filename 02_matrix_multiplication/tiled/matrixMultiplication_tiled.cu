
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include <algorithm>

using namespace std;

__global__ void matrixMultiplication(const int* matrixA, const int* matrixB,
                                     int* matrixC, int N) {

    // CUDA shared memory does not allow dynamic memory allocation
    __shared__ int sharedA[256];
    __shared__ int sharedB[256];

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;  

    int tmp = 0;

    // Loop over all shared memory tiles
    for (int k=0; k<N; k+=blockDim.x) {

        // Fill the shared memory tile
        sharedA[threadIdx.y*blockDim.x + threadIdx.x] = matrixA[row*N + k + threadIdx.x];
        sharedB[threadIdx.y*blockDim.x + threadIdx.x] = matrixB[(k + threadIdx.y)*N + col];

        // Let all threads in the block finish filling the shared memory tile
        // Acts like block level synchronization barrier
        __syncthreads();

        // Perform matmul operation using shared memory tiles
        for (int idx=0; idx<blockDim.x; idx++) {
            tmp += sharedA[threadIdx.y*blockDim.x + idx] * 
                                    sharedB[idx*blockDim.x + threadIdx.x];
        }

        __syncthreads();
    }

    matrixC[row*N + col] = tmp;
                    
}

void verify_result(const int* matrixA, const int* matrixB,
                                     int* matrixC, int N) {

    for (int i=0; i<N; i++) {
        for (int j=0; j<N; j++) {
            int curr_val = 0;
            for (int k=0; k<N; k++) {
                curr_val += matrixA[i*N + k] * matrixB[k*N + j];
            }
            assert(matrixC[i*N + j] == curr_val);
        }
    }
}

int main() {

    // 1024 rows and columns
    int N = 1 << 10;
    size_t bytes = sizeof(int) * N * N;

    int* matrixA = new int[N*N];
    int* matrixB = new int[N*N];
    int* matrixC = new int[N*N];

    for (int i=0; i<N*N; i++) {
        matrixA[i] = rand() % 100;
        matrixB[i] = rand() % 100;
        matrixC[i] = 0;
    }

    int *matrixA_gpu, *matrixB_gpu, *matrixC_gpu;

    hipMalloc(&matrixA_gpu, bytes);
    hipMalloc(&matrixB_gpu, bytes);
    hipMalloc(&matrixC_gpu, bytes);

    hipMemcpy(matrixA_gpu, matrixA, bytes, hipMemcpyHostToDevice);
    hipMemcpy(matrixB_gpu, matrixB, bytes, hipMemcpyHostToDevice);
    hipMemcpy(matrixC_gpu, matrixC, bytes, hipMemcpyHostToDevice);

    int THREADS = 16;
    int BLOCKS = (N + THREADS - 1) / THREADS;
    
    dim3 block_size (THREADS, THREADS);
    dim3 grid_size (BLOCKS, BLOCKS);

    matrixMultiplication<<<grid_size, block_size>>>(matrixA_gpu, matrixB_gpu, matrixC_gpu, N);

    hipMemcpy(matrixC, matrixC_gpu, bytes, hipMemcpyDeviceToHost);

    verify_result(matrixA, matrixB, matrixC, N);

    hipFree(matrixA_gpu);
    hipFree(matrixB_gpu);
    hipFree(matrixC_gpu);

    cout << "Run Successful" << endl;

}
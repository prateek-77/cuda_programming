
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>

using namespace std;

const int NUM_THREADS = 256;

__global__ void sum_reduction(const int* arr, int* res_arr) {

    __shared__ int shmem[NUM_THREADS];
    
    // Start performing sum reduction while filling shared memory itself.
    // This reduces the number of blocks required by half!
    // Number of threads per block still remain the same.
    // Overall, number of idle threads consumed by the program is still reduced.
    int i = blockIdx.x*blockDim.x*2 + threadIdx.x;

    // Fill shared memory with elements
    shmem[threadIdx.x] = arr[i] + arr[i + blockDim.x];
    __syncthreads();

    // Loop with largest stride, and reduce it by 2 every iteration until it reaches 1.
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        int idx = threadIdx.x;
        if (idx < s) {
            shmem[idx] += shmem[idx + s];
        }
        __syncthreads();
    }

    if (threadIdx.x==0) res_arr[blockIdx.x] = shmem[0];
}


int main() {
    
    int N = 1 << 16;
    size_t bytes = sizeof(int) * N;
    size_t bytes_res = sizeof(int) * (N / NUM_THREADS); // 2^16 split by 256 threads = 256.

    vector<int> arr;
    vector<int> result_arr;

    arr.resize(N);
    result_arr.resize(N / NUM_THREADS);

    for (int i=0; i<N; i++) {
        arr[i] = rand() % 10;
    }

    int *arr_gpu, *result_arr_gpu;

    hipMalloc(&arr_gpu, bytes);
    hipMalloc(&result_arr_gpu, bytes_res);

    hipMemcpy(arr_gpu, arr.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = NUM_THREADS;
    // Halve the number of blocks required
    int BLOCKS = (N + THREADS - 1) / THREADS / 2;

    // Reduce array from 2^16 to 2^8 (256)
    sum_reduction<<<BLOCKS, THREADS>>> (arr_gpu, result_arr_gpu);
    // Reduce above array to get result in the first index
    sum_reduction<<<1, THREADS>>> (result_arr_gpu, result_arr_gpu);

    hipMemcpy(result_arr.data(), result_arr_gpu, bytes_res, hipMemcpyDeviceToHost);
    assert(result_arr[0] == accumulate(arr.begin(), arr.end(), 0));

    cout << "Run Successful" << endl;

    hipFree(arr_gpu);
    hipFree(result_arr_gpu);
}

#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <assert.h>
#include <iostream>
#include <vector>
#include <algorithm>
#include <numeric>

using namespace std;

const int NUM_THREADS = 256;

__global__ void sum_reduction(const int* arr, int* res_arr) {

    __shared__ int shmem[NUM_THREADS];
    
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    // Fill shared memory with elements
    shmem[threadIdx.x] = arr[tid];
    __syncthreads();

    // Loop with largest stride, and reduce it by 2 every iteration until it reaches 1.
    for (int s = blockDim.x/2; s > 0; s >>= 1) {
        // Bank Conflicts are minimized here. Shmem -> memory banks. 
        // Each bank can handle one memory access per clock cycle. 
        // If multiple/same threads access the same bank at the same time, it results in a conflict.
        // This forces the hardware to serialize the accesses, which slows down performance.
        // In the new case, instead of gradually increasing the stride for memory access, 
        // we start with the highest stride, and decrease it. 
        // This way not all threads are centered around idx 0, especially in the later stages of loop.
        int idx = threadIdx.x;
        if (idx < s) {
            shmem[idx] += shmem[idx + s];
        }
        __syncthreads();
    }

    if (threadIdx.x==0) res_arr[blockIdx.x] = shmem[0];
}


int main() {
    
    int N = 1 << 16;
    size_t bytes = sizeof(int) * N;
    size_t bytes_res = sizeof(int) * (N / NUM_THREADS); // 2^16 split by 256 threads = 256.

    vector<int> arr;
    vector<int> result_arr;

    arr.resize(N);
    result_arr.resize(N / NUM_THREADS);

    for (int i=0; i<N; i++) {
        arr[i] = rand() % 10;
    }

    int *arr_gpu, *result_arr_gpu;

    hipMalloc(&arr_gpu, bytes);
    hipMalloc(&result_arr_gpu, bytes_res);

    hipMemcpy(arr_gpu, arr.data(), bytes, hipMemcpyHostToDevice);

    int THREADS = NUM_THREADS;
    int BLOCKS = (N + THREADS - 1) / THREADS;

    // Reduce array from 2^16 to 2^8 (256)
    sum_reduction<<<BLOCKS, THREADS>>> (arr_gpu, result_arr_gpu);
    // Reduce above array to get result in the first index
    sum_reduction<<<1, THREADS>>> (result_arr_gpu, result_arr_gpu);

    hipMemcpy(result_arr.data(), result_arr_gpu, bytes_res, hipMemcpyDeviceToHost);
    assert(result_arr[0] == accumulate(arr.begin(), arr.end(), 0));

    cout << "Run Successful" << endl;

    hipFree(arr_gpu);
    hipFree(result_arr_gpu);
}